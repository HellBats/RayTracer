#include "hip/hip_runtime.h"
#include "Geometry.h"
#include<iostream>


const float kepsilon = 1e-6; 

__host__ __device__ bool solveQuadratic(float a, float b,float c,float *t0,float *t1)
{   
    float discriminant  = b*b-4*a*c;
    if(discriminant<0) return false;
    *t0 = (-b + sqrt(discriminant))/(2*a); 
    *t1 = (-b - sqrt(discriminant))/(2*a);
    return true;
}


void InitalizeSphere(Sphere &sphere,float &radius, vec3 &center, vec3 &albedo)
{
    sphere.radius = radius;
    sphere.center = center;
    sphere.albedo = albedo;
}

void InitalizePlane(Plane &plane,vec3 &point, vec3 &normal, vec3 &albedo)
{
    plane.point = point;
    plane.normal = normal;
    plane.albedo = albedo;
}

void InitalizeTriangle(Triangle &triangle,TriVertices &vertices, vec3 &albedo)
{
    triangle.vertices.a = vertices.a;
    triangle.vertices.b = vertices.b;
    triangle.vertices.c = vertices.c;
    triangle.normal = cross((vertices.b-vertices.a),(vertices.c-vertices.a)); 
    triangle.origin_distance = -dot(vertices.a,triangle.normal);
    triangle.is_double_sided = true;
    triangle.albedo = albedo;
}


__host__ __device__ bool Intersect(Geometry& g,Ray& r, float& t, float& u, float& v) {
    switch (g.type) {
        case GeometryType::SPHERE:
            return IntersectSphere(g.sphere, r, t,u,v);
        case GeometryType::PLANE:
            return IntersectPlane(g.plane, r, t,u,v);
        case GeometryType::TRIANGLE:
            return IntersectTriangle(g.triangle, r, t, u, v);
    }
    return false;
}

__host__ __device__ bool IntersectSphere(Sphere &sphere,Ray& r,float& t,float &u, float &v)
{
    vec3 L = r.origin - sphere.center;
    float a = dot(r.direction,r.direction);
    float b = 2 * dot(r.direction,L);
    float c = dot(L,L) - sphere.radius * sphere.radius;
    float t0,t1;
    if (!solveQuadratic(a, b, c, &t0, &t1)) return false;
    // printf("%f, %f\n",t0,t1);
    t = t0>t1?t1:t0;
    if(t>0) 
    {
        u=2;
        v=2;
        return true;
    }
    return false;
}


__host__ __device__ bool IntersectPlane(Plane &plane ,Ray& r,float &t,float &u, float &v)
{
    float denominator = dot(r.direction,plane.normal);
    if(denominator>kepsilon)
    {
        t = dot((plane.point - r.origin),plane.normal)/denominator;
        if(t>=0)
        {
            u=2;
            v=2;
            return true;
        }
    }
    return false;
}

__host__ __device__ bool IntersectTriangle(Triangle &triangle,Ray& r,float& t, float& u, float& v)
{
    
    vec3 v0v1 = triangle.vertices.b - triangle.vertices.a;
    vec3 v0v2 = triangle.vertices.c - triangle.vertices.a;
    vec3 pvec = cross(v0v2,r.direction);
    float det = dot(pvec,v0v1);
    // If the determinant is negative, the triangle is back-facing.
    // If the determinant is close to 0, the ray misses the triangle.
    // If det is close to 0, the ray and triangle are parallel.
    if (fabs(det) < kepsilon) return false;
    float invDet = 1 / det;

    vec3 tvec = r.origin - triangle.vertices.a;
    u = dot(pvec,tvec) * invDet;
    if (u < 0 || u > 1) return false;

    vec3 qvec = cross(v0v1,tvec);
    v = dot(qvec,r.direction) * invDet;
    if (v < 0 || u + v > 1) return false;
    t = dot(qvec,v0v2) * invDet;
    return t>0;
}
