#include "hip/hip_runtime.h"
#include "Renderer.h"
#include <iostream>

// ------------------ Forward declarations ------------------
__global__ void RenderKernel(Scene* scene,unsigned char* device_buffer, uint32_t width, uint32_t height);
__host__ __device__ u8vec3 Trace(Scene* scene,Ray &r,Geometry*& hitObject,HitRecord &record);

// ----------------------------------------------------------

Renderer::Renderer(std::vector<unsigned char>& pixels,uint32_t width, uint32_t height)
:pixels(pixels), width(width), height(height)
{

}


__host__ __device__ void RenderPixel(Scene* scene, uint32_t i, uint32_t j,
                                     u8vec3 &color, int width, int height)
{
    float scale = tan(scene->camera.fov * 0.5f);
    float Px = (2 * ((i + 0.5f) / width) - 1) * scale * scene->camera.aspect_ratio;
    float Py = (1 - 2 * ((j + 0.5f) / height)) * scale;
    Geometry* hitObject = nullptr;

    // Recompute transformation (world matrix of the camera)
    InitializeTransformation(&scene->camera);
    // printf("%d",scene->object_count);
    // Ray origin = camera position in world space
    vec3 rayOriginWorld = scene->camera.position;
    // Pixel point in camera space (on near plane z=-1)
    vec4 pixelCam = {Px, Py, -1, 0};  // direction, w=0
    // Rotate into world space using camera transform
    pixelCam = scene->camera.transformation * pixelCam;
    vec3 rayDirWorld = normalize(vec3{pixelCam.x,pixelCam.y,pixelCam.z});
    Ray ray{rayOriginWorld, rayDirWorld};
    HitRecord record;
    record.u=2;
    record.v=2;
    color = Trace(scene, ray, hitObject,record);
}

void Renderer::RenderCPU(Scene &scene)
{
    // std::cout<<pixels.size()<<std::endl;
    for(int j=0;j<height;j++)
    {
        for(int i=0;i<width;i++)
        {
            u8vec3 colors; 
            RenderPixel(&scene,i,j,colors,width,height);
            int idx = (j*width+i)*4;
            pixels[idx + 0] = colors.r;
            pixels[idx + 1] = colors.g;
            pixels[idx + 2] = colors.b;
            pixels[idx + 3] = 255;
        }
    }
    return ;
}

void Renderer::RenderGPU(Scene &scene)
{
    dim3 block(16,16);
    dim3 grid((width+block.x-1)/block.x, (height+block.y-1)/block.y);
    // -------- Allocate GPU buffer --------
    unsigned char* device_buffer;
    hipMalloc(&device_buffer, sizeof(unsigned char) * width * height * 4);

    // -------- Copy objects --------
    Geometry* d_objects;
    hipMalloc(&d_objects, sizeof(Geometry) * scene.object_count);
    hipMemcpy(d_objects, scene.objects,
               sizeof(Geometry) * scene.object_count,
               hipMemcpyHostToDevice);


    // -------- Copy lights --------
    Light* d_lights;
    hipMalloc(&d_lights, sizeof(Light) * scene.lights_count);
    hipMemcpy(d_lights, scene.lights,
               sizeof(Light) * scene.lights_count,
               hipMemcpyHostToDevice);

    

    // -------- Prepare patched Scene --------
    Scene scene_copy = scene;        // copy original
    scene_copy.objects = d_objects;  // patch objects pointer
    scene_copy.lights = d_lights;   // patch lights pointer

    // -------- Copy Scene to device --------
    Scene* d_scene;
    hipMalloc(&d_scene, sizeof(Scene));
    hipMemcpy(d_scene, &scene_copy, sizeof(Scene), hipMemcpyHostToDevice);
    // printf("Host scene objects = %d\n", scene.object_count);
    // -------- Launch kernel --------
    RenderKernel<<<grid, block>>>(d_scene, device_buffer, width, height);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    // -------- Copy back pixels --------
    hipMemcpy(pixels.data(), device_buffer,
               width * height * 4 * sizeof(unsigned char),
               hipMemcpyDeviceToHost);

    // -------- Cleanup --------
    hipFree(d_objects);
    hipFree(d_scene);
    hipFree(device_buffer);
}

__global__ void RenderKernel(Scene* scene,unsigned char* device_buffer, uint32_t width, uint32_t height)
{
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    if (x >= width || y >= height) return;
    u8vec3 colors;
    RenderPixel(scene,x,y,colors,width,height);

    int idx = (y * width + x) * 4;
    device_buffer[idx + 0] = colors.r;
    device_buffer[idx + 1] = colors.g;
    device_buffer[idx + 2] = colors.b;
    device_buffer[idx + 3] = 255;
}


__host__ __device__ u8vec3 Trace(Scene* scene,Ray &r,Geometry*& hitObject,HitRecord &record)
{
    record.t = std::numeric_limits<float>::max();
    u8vec3 color;
    for(int d=0;d<1;d++)
    {
        for (int i=0;i<scene->object_count;i++) {
            float t = record.t; 
            if (Intersect(scene->objects[i],r, record) && record.t < t) {
                hitObject = &scene->objects[i];
                record.material = hitObject->material;
                t = record.t;
            }
        }
        color = Shade(hitObject,scene->lights,r, record,scene->lights_count);
    }
    return color;
}
