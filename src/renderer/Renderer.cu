#include "hip/hip_runtime.h"
#include "Renderer.h"
#include <iostream>

// ------------------ Forward declarations ------------------
__global__ void RenderKernel(Scene* scene,unsigned char* device_buffer, uint32_t width, uint32_t height);
__host__ __device__ u8vec3 Trace(Scene* scene,Ray &r);

// ----------------------------------------------------------

Renderer::Renderer(std::vector<unsigned char>& pixels,uint32_t width, uint32_t height)
:pixels(pixels), width(width), height(height)
{

}


__host__ __device__ void RenderPixel(Scene* scene, uint32_t i, uint32_t j,
                                     u8vec3 &color, int width, int height)
{
    float scale = tan(scene->camera.fov * 0.5f);
    float Px = (2 * ((i + 0.5f) / width) - 1) * scale * scene->camera.aspect_ratio;
    float Py = (1 - 2 * ((j + 0.5f) / height)) * scale;

    // Recompute transformation (world matrix of the camera)
    InitializeTransformation(&scene->camera);
    // printf("%d",scene->object_count);
    // Ray origin = camera position in world space
    vec3 rayOriginWorld = scene->camera.position;
    // Pixel point in camera space (on near plane z=-1)
    vec4 pixelCam = {Px, Py, -1, 0};  // direction, w=0
    // Rotate into world space using camera transform
    pixelCam = scene->camera.transformation * pixelCam;
    vec3 rayDirWorld = normalize(vec3{pixelCam.x,pixelCam.y,pixelCam.z});
    Ray ray{.type = RayType::PrimaryRay,.origin = rayOriginWorld,.direction = rayDirWorld};
    color = Trace(scene, ray);
}

void Renderer::RenderCPU(Scene &scene)
{
    // std::cout<<pixels.size()<<std::endl;
    for(int j=0;j<height;j++)
    {
        for(int i=0;i<width;i++)
        {
            u8vec3 colors; 
            RenderPixel(&scene,i,j,colors,width,height);
            int idx = (j*width+i)*4;
            pixels[idx + 0] = colors.r;
            pixels[idx + 1] = colors.g;
            pixels[idx + 2] = colors.b;
            pixels[idx + 3] = 255;
        }
    }
    return ;
}

void Renderer::RenderGPU(Scene &scene)
{
    dim3 block(16,16);
    dim3 grid((width+block.x-1)/block.x, (height+block.y-1)/block.y);
    // -------- Allocate GPU buffer --------
    unsigned char* device_buffer;
    hipMalloc(&device_buffer, sizeof(unsigned char) * width * height * 4);

    // -------- Copy objects --------
    Geometry* d_objects;
    hipMalloc(&d_objects, sizeof(Geometry) * scene.object_count);
    hipMemcpy(d_objects, scene.objects,
               sizeof(Geometry) * scene.object_count,
               hipMemcpyHostToDevice);


    // -------- Copy lights --------
    Light* d_lights;
    hipMalloc(&d_lights, sizeof(Light) * scene.lights_count);
    hipMemcpy(d_lights, scene.lights,
               sizeof(Light) * scene.lights_count,
               hipMemcpyHostToDevice);

    

    // -------- Prepare patched Scene --------
    Scene scene_copy = scene;        // copy original
    scene_copy.objects = d_objects;  // patch objects pointer
    scene_copy.lights = d_lights;   // patch lights pointer

    // -------- Copy Scene to device --------
    Scene* d_scene;
    hipMalloc(&d_scene, sizeof(Scene));
    hipMemcpy(d_scene, &scene_copy, sizeof(Scene), hipMemcpyHostToDevice);
    // printf("Host scene objects = %d\n", scene.object_count);
    // -------- Launch kernel --------
    RenderKernel<<<grid, block>>>(d_scene, device_buffer, width, height);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    // -------- Copy back pixels --------
    hipMemcpy(pixels.data(), device_buffer,
               width * height * 4 * sizeof(unsigned char),
               hipMemcpyDeviceToHost);

    // -------- Cleanup --------
    hipFree(d_objects);
    hipFree(d_scene);
    hipFree(device_buffer);
}

__global__ void RenderKernel(Scene* scene,unsigned char* device_buffer, uint32_t width, uint32_t height)
{
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    if (x >= width || y >= height) return;
    u8vec3 colors;
    RenderPixel(scene,x,y,colors,width,height);

    int idx = (y * width + x) * 4;
    device_buffer[idx + 0] = colors.r;
    device_buffer[idx + 1] = colors.g;
    device_buffer[idx + 2] = colors.b;
    device_buffer[idx + 3] = 255;
}


__host__ __device__ vec3 Background(Ray& r) {
    vec3 unit_dir = normalize(r.direction);
    float t = 0.5f * (unit_dir.y + 1.0f); // map y from [-1,1] to [0,1]
    vec3 color = (1.0f - t) * vec3{1.0, 1.0, 1.0} + t * vec3{0.5, 0.7, 1.0};
    return color;
}

__host__ __device__ u8vec3 Trace(Scene* scene,Ray &r)
{
    vec3 background_color = Background(r);
    size_t MAX_DEPTH = 5;
    int depth = MAX_DEPTH;
    HitStack stack;
    vec3 color = background_color;
    float bias = 1e-4;
    stack.PushRay(r);
    int counter = scene->lights_count-1;
    while(!stack.RayIsEmpty())
    {
        Ray new_ray = stack.RayPop();
        HitRecord record;
        FillIntersectionRecord(scene,new_ray,record);
        if(record.t==std::numeric_limits<float>::max() && new_ray.type==RayType::PrimaryRay)
        {
            return convert_to_u8vec3(background_color*255);
        }
        if(new_ray.type==RayType::PrimaryRay || new_ray.type==RayType::ReflectionRay)
        {
            HitRecord old_record = stack.RecordTop();
            for(int i=0;i<scene->lights_count;i++)
            {
                vec3 lightDir = GetLightDirection(scene->lights[i],record.intersection);
                Ray next_ray = Ray{.type=RayType::ShadowRay,.origin=record.intersection+record.normal*bias,
                    .direction=lightDir};
                stack.PushRay(next_ray);
            }
            if(record.material.reflectivity>0 && depth>0)
            {
                Ray next_ray = Ray{.type=RayType::ReflectionRay,.origin=record.intersection+record.normal*bias,
                    .direction=reflect(new_ray.direction,record.normal)};
                stack.PushRay(next_ray);
            }
            stack.PushRecord(record);
            depth--;
        }
        else
        {
            FillIntersectionRecord(scene,new_ray,record);
            HitRecord old_record = stack.RecordTop();

            vec3 localColor = vec3{0,0,0};   // diffuse shading accumulator
            Shade(scene->lights[counter], new_ray, record, old_record, localColor);
            counter--;

            if(counter == -1)  
            {
                // Here 'color' is actually the reflection contribution returned
                vec3 reflectionColor = color;  

                // Combine reflection and local shading
                color = (1 - old_record.material.reflectivity) * localColor 
                    + old_record.material.reflectivity * reflectionColor;

                counter = scene->lights_count - 1;
                stack.RecordPop();
            }
        }
    }
    color.x = fminf(color.x, 1.0f);
    color.y = fminf(color.y, 1.0f);
    color.z = fminf(color.z, 1.0f);
    return convert_to_u8vec3(color*255);
}


__host__ __device__ void FillIntersectionRecord(Scene* scene,Ray &r, HitRecord &record)
{
    HitRecord nearest;
    Geometry hitObject;
    record.u=2;
    record.v=2;
    record.ray_direction = r.direction; 
    record.t = std::numeric_limits<float>::max();
    nearest = record;
    for (int i=0;i<scene->object_count;i++) 
    {
        if (Intersect(scene->objects[i],r, nearest) && record.t > nearest.t)
        {
            hitObject = scene->objects[i];
            record = nearest;
            record.material = hitObject.material;
        }
    }
}
